// compile : nvcc -std=c++11 22111001-prob3-cudamalloc.cu -o prob3-1
// Execute : ./prob3-1

#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

const uint64_t N = (1 << 10);

using std::cerr;
using std::cout;
using std::endl;

__global__ void opt_matmul_prob2(const uint64_t* d_A, const uint64_t* d_B, uint64_t* d_C) {
  // TODO: Fill in
  const int width = 16;
  __shared__ uint64_t left[width][width];
  __shared__ uint64_t right[width][width];

  uint64_t i = blockIdx.y * blockDim.y + threadIdx.y;
  uint64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  if((i < N) && (j < N)){
    uint64_t x = 0;
    for(int k = 0; k < (N/width); k++){
      left[threadIdx.y][threadIdx.x] = d_A[i*N + k*width + threadIdx.x];
      right[threadIdx.y][threadIdx.x] = d_B[(k * width + threadIdx.y) * N + j];
      __syncthreads();
      for(int z = 0; z<width; z++){
        x += left[threadIdx.y][z] * right[z][threadIdx.x];
      __syncthreads();

      }
    }
    d_C[i*N + j] = x;
  }
}

__host__ void cpumatMul(const uint64_t* h_A, const uint64_t* h_B, uint64_t* h_C) {
  for (uint64_t i = 0; i < N; i++) {
    for (uint64_t j = 0; j < N; j++) {
      float sum = 0.0;
      for (uint64_t k = 0; k < N; k++) {
        sum += h_A[i * N + k] * h_B[k * N + j];
      }
      h_C[i * N + j] = sum;
    }
  }
}

__host__ void check_result(const uint64_t* w_ref, const uint64_t* w_opt) {
  bool wrong = false;
  for (uint64_t i = 0; i < N; i++) {
    for (uint64_t j = 0; j < N; j++) {
      if (w_ref[i * N + j] != w_opt[i * N + j]) {
        wrong = true;
        goto out;
      }
    }
  }
out:
  if (wrong) {
    cout << " Diffs found!" << endl;
  } else {
    cout << "No differences found between base and test versions\n";
  }
}


double rtclock() { // Seconds
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) {
    cout << "Error return from gettimeofday: " << stat << "\n";
  }
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

int main() {
  const uint64_t SIZE = N * N;

  uint64_t *h_A, *h_B, *h_cpu_C, *h_gpu1_C;

  h_A = (uint64_t*)malloc(SIZE * sizeof(uint64_t));
  h_B = (uint64_t*)malloc(SIZE * sizeof(uint64_t));
  h_cpu_C = (uint64_t*)malloc(SIZE * sizeof(uint64_t));
  h_gpu1_C = (uint64_t*)malloc(SIZE * sizeof(uint64_t));

  for (uint64_t i = 0; i < N; i++) {
    for (uint64_t j = 0; j < N; j++) {
      h_A[i * N + j] = rand() % 64;
      h_B[i * N + j] = 2;
      h_cpu_C[i * N + j] = 0;
      h_gpu1_C[i * N + j] = 0;
    }
  }

  double clkbegin = rtclock();
  cpumatMul(h_A, h_B, h_cpu_C);
  double clkend = rtclock();
  double cpu_time = clkend - clkbegin;
  cout << "Matmul time on CPU: " << cpu_time * 1000 << " msec\n" << endl;

  hipError_t status;
  hipEvent_t start, end;

  uint64_t *d_A, *d_B, *d_C1;
  status = hipMalloc(&d_A, SIZE * sizeof(uint64_t));
  if (status != hipSuccess) {
    cerr << hipGetErrorString(status) << endl;
  }
  status = hipMalloc(&d_B, SIZE * sizeof(uint64_t));
  if (status != hipSuccess) {
    cerr << hipGetErrorString(status) << endl;
  }
  status = hipMalloc(&d_C1, SIZE * sizeof(uint64_t));
  if (status != hipSuccess) {
    cerr << hipGetErrorString(status) << endl;
  }

  hipEventCreate(&start);
  hipEventCreate(&end);
  
  float k1_htd_time, k1_kernel_time, k1_dth_time;

  // Host To Device
  hipEventRecord(start);
  status = hipMemcpy(d_A, h_A, SIZE * sizeof(uint64_t), hipMemcpyHostToDevice);
  if (status != hipSuccess) {
    cerr << hipGetErrorString(status) << endl;
  }
  status = hipMemcpy(d_B, h_B, SIZE * sizeof(uint64_t), hipMemcpyHostToDevice);
  if (status != hipSuccess) {
    cerr << hipGetErrorString(status) << endl;
  }
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&k1_htd_time, start, end);

  // Kernel
  hipEventRecord(start);
  dim3 grid((1<<6), (1<<6), 1);
  dim3 block((1<<4), (1<<4), 1);
  opt_matmul_prob2<<<grid, block>>>(d_A, d_B, d_C1);
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&k1_kernel_time, start, end);
  
  // Device to Host
  hipEventRecord(start);
  hipMemcpy(h_gpu1_C, d_C1, SIZE * sizeof(uint64_t), hipMemcpyDeviceToHost);
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&k1_dth_time, start, end);

  hipEventDestroy(start);
  hipEventDestroy(end);
  
  // Checking result
  check_result(h_cpu_C, h_gpu1_C);
  cout << "Timings for kernel 1 :\n";
  cout << "Host To Device Copy(ms) : " << k1_htd_time << endl;
  cout << "Kernel(ms) : " << k1_kernel_time << endl;
  cout << "Device To Host Copy(ms) : " << k1_dth_time << endl << endl;
  
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C1);
  
  free(h_A);
  free(h_B);
  free(h_cpu_C);
  free(h_gpu1_C);

  return EXIT_SUCCESS;
}
