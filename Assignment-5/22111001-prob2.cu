// compile : nvcc -std=c++11 22111001-prob2.cu -o prob2
// Execute : ./prob2

#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

const uint64_t N = (1 << 10);

using std::cerr;
using std::cout;
using std::endl;

__global__ void kernel1(const uint64_t* d_A, const uint64_t* d_B, uint64_t* d_C) {
  // There will be 1<<14 number of blocks with dimension (1<<7, 1<<7, 1)
  // Every block will have 1024 threads in it with dimension (1<<6, 1<<4)
  uint64_t i = blockIdx.y * blockDim.y + threadIdx.y;
  uint64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  if((i < N) && (j < N)){
    uint64_t x = 0;
    for(int k = 0; k<N; k++){
      x += d_A[i*N + k] * d_B[k*N + j];
    }
    d_C[i*N + j] = x;
  }
}

__global__ void kernel2(const uint64_t* d_A, const uint64_t* d_B, uint64_t* d_C) {
  // TODO: Fill in
  const int width = 32;
  __shared__ uint64_t left[width][width];
  __shared__ uint64_t right[width][width];

  uint64_t i = blockIdx.y * blockDim.y + threadIdx.y;
  uint64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  if((i < N) && (j < N)){
    uint64_t x = 0;
    for(int k = 0; k < (N/width); k++){
      left[threadIdx.y][threadIdx.x] = d_A[i*N + k*width + threadIdx.x];
      right[threadIdx.y][threadIdx.x] = d_B[(k * width + threadIdx.y) * N + j];
      __syncthreads();
      for(int z = 0; z<width; z++){
        x += left[threadIdx.y][z] * right[z][threadIdx.x];
      __syncthreads();

      }
    }
    d_C[i*N + j] = x;
  }
}

__host__ void cpumatMul(const uint64_t* h_A, const uint64_t* h_B, uint64_t* h_C) {
  for (uint64_t i = 0; i < N; i++) {
    for (uint64_t j = 0; j < N; j++) {
      float sum = 0.0;
      for (uint64_t k = 0; k < N; k++) {
        sum += h_A[i * N + k] * h_B[k * N + j];
      }
      h_C[i * N + j] = sum;
    }
  }
}

__host__ void check_result(const uint64_t* w_ref, const uint64_t* w_opt) {
  bool wrong = false;
  for (uint64_t i = 0; i < N; i++) {
    for (uint64_t j = 0; j < N; j++) {
      if (w_ref[i * N + j] != w_opt[i * N + j]) {
        wrong = true;
        goto out;
      }
    }
  }
out:
  if (wrong) {
    cout << "Diffs found!" << endl;
  } else {
    cout << "No differences found between base and test versions\n";
  }
}

double rtclock() { // Seconds
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) {
    cout << "Error return from gettimeofday: " << stat << "\n";
  }
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

int main() {
  const uint64_t SIZE = N * N;

  uint64_t *h_A, *h_B, *h_cpu_C, *h_gpu1_C, *h_gpu2_C;

  h_A = (uint64_t*)malloc(SIZE * sizeof(uint64_t));
  h_B = (uint64_t*)malloc(SIZE * sizeof(uint64_t));
  h_cpu_C = (uint64_t*)malloc(SIZE * sizeof(uint64_t));
  h_gpu1_C = (uint64_t*)malloc(SIZE * sizeof(uint64_t));
  h_gpu2_C = (uint64_t*)malloc(SIZE * sizeof(uint64_t));

  for (uint64_t i = 0; i < N; i++) {
    for (uint64_t j = 0; j < N; j++) {
      h_A[i * N + j] = rand() % 64;
      h_B[i * N + j] = 2;
      h_cpu_C[i * N + j] = 0;
      h_gpu1_C[i * N + j] = 0;
      h_gpu2_C[i * N + j] = 0;
    }
  }

  double clkbegin = rtclock();
  cpumatMul(h_A, h_B, h_cpu_C);
  double clkend = rtclock();
  double cpu_time = clkend - clkbegin;
  cout << "Matmul time on CPU: " << cpu_time * 1000 << " msec" << endl;

  hipError_t status;
  hipEvent_t start, end;
  
  uint64_t *d_A, *d_B, *d_C1;
  status = hipMalloc(&d_A, SIZE * sizeof(uint64_t));
  if (status != hipSuccess) {
    cerr << hipGetErrorString(status) << endl;
  }
  status = hipMalloc(&d_B, SIZE * sizeof(uint64_t));
  status = hipMalloc(&d_C1, SIZE * sizeof(uint64_t));
  status = hipMemcpy(d_A, h_A, SIZE * sizeof(uint64_t), hipMemcpyHostToDevice);
  status = hipMemcpy(d_B, h_B, SIZE * sizeof(uint64_t), hipMemcpyHostToDevice);
  
  dim3 grid((1<<5), (1<<5), 1);
  dim3 block((1<<5), (1<<5), 1);
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);
  kernel1<<<grid, block>>>(d_A, d_B, d_C1);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float kernel_time;
  hipEventElapsedTime(&kernel_time, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);
  hipMemcpy(h_gpu1_C, d_C1, SIZE * sizeof(uint64_t), hipMemcpyDeviceToHost);
  check_result(h_cpu_C, h_gpu1_C);
  std::cout << "Kernel 1 time (ms): " << kernel_time << "\n";


  uint64_t* d_C2;
  status = hipMalloc(&d_C2, SIZE * sizeof(uint64_t));
  float ti;
  dim3 grid2((1<<5), (1<<5), 1);
  dim3 block2((1<<5), (1<<5), 1);
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);
  kernel2<<<grid2, block2>>>(d_A, d_B, d_C2);
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&ti, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);
  hipMemcpy(h_gpu2_C, d_C2, SIZE * sizeof(uint64_t), hipMemcpyDeviceToHost);
  check_result(h_cpu_C, h_gpu2_C);
  std::cout << "Kernel 2 time (ms): " << ti << "\n";


  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C1);
  hipFree(d_C2);

  free(h_A);
  free(h_B);
  free(h_cpu_C);
  free(h_gpu1_C);
  free(h_gpu2_C);

  return EXIT_SUCCESS;
}
